﻿#include "network_gpu.h"

const string DATA_PATH = "C:/Users/semio/Downloads/chess_data1.bin";
const int EPOCHS = 30;

vector<string> TEST_POSITIONS = {
    "3k4/5K2/8/4P3/8/8/8/8 b - - 2 11", // KPvk
    "4k3/8/3KP3/8/8/8/8/8 b - - 0 10", // KPvk
    "rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w KQkq - 0 1", // start position
    "2k2bnr/ppp2ppp/2n5/3rP3/4pB2/8/PPP1KPPP/R5NR w - - 0 10", // random position (black is winning)
    "1k5r/p1pR3p/2p3pb/4Pp2/4p3/8/PPP1KPPP/7R b - - 1 16", // random position (white is winning)
    "7Q/6K1/p7/2p1Q3/8/8/P1P5/2k5 b - - 4 57", // mate in 3 position (white to mate)
    "r5k1/p5p1/4p2p/4p3/3p4/P6K/5q2/8 b - - 0 33" // mate in 3 position (black to mate)
};

vector<float> TEST_POSITIONS_EVAL = {
    114, 0, 0.2f, -5.5f, 4.1f, 122, -122
};


int main()
{
    // LOAD AND NORMALIZE DATA
    vector<NetInput> DATA = getNetData(DATA_PATH, 100000);

    vector<float> targets;
    for (auto& i : DATA) {
        targets.push_back(i.target);
    }

    vector<float> targetsNormalized = normalizeTargets(targets);
    for (int i = 0; i < DATA.size(); ++i) {
        DATA[i].target = targetsNormalized[i];
    }

    // TRAIN NETWORK
    Network_GPU net("");
    net.train(DATA, EPOCHS);

    // TEST NEURAL NETWORK
    for (int i = 0; i < TEST_POSITIONS.size(); ++i) {
        float prediction = net.evaluatePosition(TEST_POSITIONS[i]);
        cout << "Fen: " << TEST_POSITIONS[i] << endl;
        cout << "Prediction: " << prediction << endl;
        cout << "Target: " << TEST_POSITIONS_EVAL[i] << endl;
        cout << "--------------------------------------" << endl;
    }

    return 0;
}
